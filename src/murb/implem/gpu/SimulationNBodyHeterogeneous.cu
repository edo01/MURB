#include "hip/hip_runtime.h"
#include <cassert>
#include <cmath>
#include <fstream>
#include <iostream>
#include <limits>
#include <string>

#include <hip/hip_runtime.h>
#include <>

#include "SimulationNBodyHeterogeneous.hpp"
#include "commons.cuh"

/**
 * Here all the computation depends on the size of the tile. In order to work the tile size must be greater than the 
 * number of threads in the block.
 */
__global__ void
calculate_forces_no_border(float* devX, float* devY, float* devZ, float* devM, const unsigned long N, 
                 const float softSquared, const float G, float* devAx, float* devAy, float* devAz,
                 const float TILE_SIZE)
{
    extern __shared__ float4 shPosition[]; // shared memory to store the positions of the bodies
    int j, tile;
    float3 acc = {0.0f, 0.0f, 0.0f}; // acceleration of the body
    int id_i = blockIdx.x * blockDim.x + threadIdx.x; // id of the i-th resident body 
	
	// load the resident body
    float4 bi = make_float4(devX[id_i], devY[id_i], devZ[id_i], devM[id_i]); // position and mass of the i-th

    for (j = 0, tile = 0; j < N; j += TILE_SIZE, tile++) {
		int id_j = tile * blockDim.x + threadIdx.x; 
		shPosition[threadIdx.x] = make_float4(devX[id_j], devY[id_j], devZ[id_j], id_j >= N? 0:devM[id_j]);
		__syncthreads(); // wait for all the threads to load the data
		tile_calculation(bi, &acc, G, softSquared);
		__syncthreads(); // wait before overwriting the shared memory
    }

    // save the results in the global memory
    devAx[id_i] = acc.x;
    devAy[id_i] = acc.y;
    devAz[id_i] = acc.z;
}

SimulationNBodyHeterogeneous::SimulationNBodyHeterogeneous(const unsigned long nBodies, const std::string &scheme, const float soft,
                                           const unsigned long randInit)
    : SimulationNBodyInterface(nBodies, scheme, soft, randInit)
{
    const unsigned long N = this->getBodies().getN();

    this->NTPB = 256;
    this->N_x = N - (N % this->NTPB); // vertical size of the grid
    this->N_y = N; // horizontal size of the grid
    this->N_res = N % this->NTPB;
    this->NB = (N + this->NTPB - 1) / this->NTPB;
    
	// allocate memory for the bodies
	hipMalloc(&this->d_qx, N_y * sizeof(float));
	hipMalloc(&this->d_qy, N_y * sizeof(float));
	hipMalloc(&this->d_qz, N_y * sizeof(float));
	hipMalloc(&this->d_m,  N_y * sizeof(float));

	// allocate memory for the accelerations
	hipMalloc(&this->d_ax, N_x * sizeof(float));
	hipMalloc(&this->d_ay, N_x * sizeof(float));
	hipMalloc(&this->d_az, N_x * sizeof(float));

    // allocate pinned memory for the bodies
    hipHostMalloc(&this->p_ax, N_x * sizeof(float));
    hipHostMalloc(&this->p_ay, N_x * sizeof(float));
    hipHostMalloc(&this->p_az, N_x * sizeof(float));
    
    this->accelerations.ax.resize(this->getBodies().getN());
    this->accelerations.ay.resize(this->getBodies().getN());
    this->accelerations.az.resize(this->getBodies().getN());

}

/**
 * The first approach is to remove the borders.
 */
void SimulationNBodyHeterogeneous::computeOneIteration()
{   
    const unsigned long N = this->getBodies().getN();
    const float softSquared =  this->soft*this->soft;

    // alias
    const dataSoA_t<float> &d = this->getBodies().getDataSoA();
    std::vector<float> &ax = this->accelerations.ax;
    std::vector<float> &ay = this->accelerations.ay;
    std::vector<float> &az = this->accelerations.az;

    hipMemcpy(this->d_qx, d.qx.data(), N_y * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(this->d_qy, d.qy.data(), N_y * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(this->d_qz, d.qz.data(), N_y * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(this->d_m, d.m.data(),   N_y * sizeof(float), hipMemcpyHostToDevice);

    calculate_forces_no_border<<<NB, NTPB, NTPB * sizeof(float4)>>>(this->d_qx, this->d_qy, this->d_qz, this->d_m, N_y, 
                                    softSquared, this->G, this->d_ax, this->d_ay, 
                                    this->d_az, this->NTPB);

    hipMemcpyAsync(this->p_ax, this->d_ax, N_x * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpyAsync(this->p_ay, this->d_ay, N_x * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpyAsync(this->p_az, this->d_az, N_x * sizeof(float), hipMemcpyDeviceToHost);

    computeOneIterationCPU();    

    // synchronize cpu and gpu here
    hipDeviceSynchronize();
    
    // copy the results from the pinned memory to the host memory
    for (unsigned long i = 0; i < N_x; i++) {
        ax[i] = p_ax[i];
        ay[i] = p_ay[i];
        az[i] = p_az[i];
    }

    // time integration
    this->bodies.updatePositionsAndVelocities(this->accelerations, this->dt);
}

// destructor
SimulationNBodyHeterogeneous::~SimulationNBodyHeterogeneous()
{
    // free bodies memory
    hipHostFree(this->d_qx);
    hipHostFree(this->d_qy);
    hipHostFree(this->d_qz);
    hipHostFree(this->d_m);

    hipHostFree(this->d_ax);
    hipHostFree(this->d_ay);
    hipHostFree(this->d_az);
}